#include "hip/hip_runtime.h"
#include <iostream>
#include <arrayfire.h>

#define DIMENSION 1000 // Number of pixels of the image

using namespace af;

__device__ int julia(int x, int y)
{
	hipFloatComplex constant;
	constant.x = -0.8;
	constant.y = 0.156;

	hipFloatComplex pt;
	
	float multiplier = 1.5; // Zoom factor

	// Calculate the point in complex plane corresponding to the x and y index
	float realPart = (multiplier * (float)(DIMENSION/2 - x) / (float)(DIMENSION/2));
	float complexPart = (multiplier * (float)(DIMENSION/2 - y) / (float)(DIMENSION/2));

	pt.x = realPart;
	pt.y = complexPart;

	for(int i = 0; i<200; i++)
	{
		pt = hipCmulf(pt,pt);
		pt = hipCaddf(pt,constant);    
		
		// Return 0 if point falls outside Julia set
		if(((pt.x*pt.x)+(pt.y*pt.y)) > 1000)		
			return 0;
	}

	// Return 1 if point falls inside Julia set
	return 1;
}

__global__ void checkJulia(int *result)
{
	int x_index = blockIdx.x * blockDim.x + threadIdx.x;
	int y_index = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = (y_index * gridDim.x * blockDim.x ) + x_index;

	if(offset<(DIMENSION*DIMENSION))
	{
		// Sets result[offset] to 1 if point is in Julia set else sets it to 0
		result[offset] = julia(x_index, y_index);
	}
}

int main()
{
	int *h_result = (int*)malloc(sizeof(int) * DIMENSION * DIMENSION);
	int *d_result; // Holds 0 or 1 depending on weather the correspoding point is in Julia Set
	hipMalloc((void**)&d_result, sizeof(int) * DIMENSION * DIMENSION);

	dim3 blockDim;
	blockDim.x = 8;
	blockDim.y = 8;

	dim3 gridDim;
	gridDim.x = DIMENSION/blockDim.x;
	gridDim.y = DIMENSION/blockDim.y;

	if(DIMENSION%blockDim.x != 0)
		gridDim.x++;

	if(DIMENSION%blockDim.y != 0)
		gridDim.y++;

	// Check for each pixel if it is in the Julia Set
	checkJulia<<<gridDim,blockDim>>>(d_result);
	hipDeviceSynchronize();
	hipMemcpy(h_result, d_result, sizeof(int) * DIMENSION * DIMENSION, hipMemcpyDeviceToHost);
	hipFree(d_result);
	array img(DIMENSION, DIMENSION, 3);

	// Sets the point to blue color if point is in Julia Set, else sets it to black
	for(int i = 0; i<DIMENSION; i++)
	{
		for(int j = 0; j<DIMENSION; j++)
		{
			int offset = i*DIMENSION+j;
			img(i, j, 0) = 0;
			img(i, j, 1) = 0;
			img(i, j, 2) = 255 * h_result[offset];
		}
	}
	image(img);

	std::cin.get();
	return 0;
}
